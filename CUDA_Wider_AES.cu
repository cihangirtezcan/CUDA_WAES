#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <ctime>
#include <hip/hip_runtime.h>
#include <>
#include "CUDA_Wider_AES.h"
__global__ void AES128_Exhaustive_Search(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4G, u32* rconG, u64* range, u8* SAES) {
	u64 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;
	//	int warpThreadIndexSBox = warpThreadIndex % S_BOX_BANK_SIZE;
		// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];
	//	__shared__ u32 t4S[TABLE_SIZE][S_BOX_BANK_SIZE];
	__shared__ u8 Sbox[64][32][4];

	if (threadIdx.x < TABLE_SIZE) {
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
			Sbox[threadIdx.x / 4][bankIndex][threadIdx.x % 4] = SAES[threadIdx.x];
		}
		//		for (u8 bankIndex = 0; bankIndex < S_BOX_BANK_SIZE; bankIndex++) {			t4S[threadIdx.x][bankIndex] = t4G[threadIdx.x];		}
		//		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) { Sbox[threadIdx.x/4][bankIndex][threadIdx.x % 4] = SAES[threadIdx.x]; }
		if (threadIdx.x < U32_SIZE) { ctS[threadIdx.x] = ct[threadIdx.x]; }
		if (threadIdx.x < RCON_SIZE) { rconS[threadIdx.x] = rconG[threadIdx.x]; }

	}	// </SHARED MEMORY>
	__syncthreads(); // Wait until every thread is ready
	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];	rk1Init = rk[1];	rk2Init = rk[2];	rk3Init = rk[3];
	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];	pt1Init = pt[1];	pt2Init = pt[2];	pt3Init = pt[3];
	u64 threadRange = *range;
	u64 threadRangeStart = threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / (u64)MAX_U32;
	rk3Init = rk3Init + threadRangeStart % (u64)MAX_U32;
	for (u64 rangeCount = 0; rangeCount < threadRange; rangeCount++) {
		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;		rk1 = rk1Init;		rk2 = rk2Init;		rk3 = rk3Init;
		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;		s1 = pt1Init;		s2 = pt2Init;		s3 = pt3Init;
		// First round just XORs input with key.
		s0 = s0 ^ rk0;		s1 = s1 ^ rk1;		s2 = s2 ^ rk2;		s3 = s3 ^ rk3;
		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {
			// Calculate round key
			u32 temp = rk3;
			rk0 = rk0 ^
				arithmeticRightShift((u32)Sbox[((temp >> 16) & 0xff) / 4][warpThreadIndex][((temp >> 16)) % 4], 8) ^
				arithmeticRightShift((u32)Sbox[((temp >> 8) & 0xff) / 4][warpThreadIndex][((temp >> 8)) % 4], 16) ^
				arithmeticRightShift((u32)Sbox[((temp) & 0xff) / 4][warpThreadIndex][((temp)) % 4], 24) ^
				((u32)Sbox[((temp >> 24) / 4)][warpThreadIndex][((temp >> 24) % 4)]) ^
				rconS[roundCount];
			rk1 = rk1 ^ rk0;			rk2 = rk2 ^ rk1;			rk3 = rk2 ^ rk3;
			// Table based round function
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s3 & 0xFF][warpThreadIndex], 24) ^ rk0;
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s0 & 0xFF][warpThreadIndex], 24) ^ rk1;
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s1 & 0xFF][warpThreadIndex], 24) ^ rk2;
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s2 & 0xFF][warpThreadIndex], 24) ^ rk3;
			s0 = t0;			s1 = t1;			s2 = t2;			s3 = t3;
		}
		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^
			arithmeticRightShift((u32)Sbox[((temp >> 16) & 0xff) / 4][warpThreadIndex][((temp >> 16)) % 4], 8) ^
			arithmeticRightShift((u32)Sbox[((temp >> 8) & 0xff) / 4][warpThreadIndex][((temp >> 8)) % 4], 16) ^
			arithmeticRightShift((u32)Sbox[((temp) & 0xff) / 4][warpThreadIndex][((temp)) % 4], 24) ^
			((u32)Sbox[((temp >> 24) / 4)][warpThreadIndex][((temp >> 24) % 4)]) ^
			rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = arithmeticRightShift((u32)Sbox[((t0 >> 24)) / 4][warpThreadIndex][((t0 >> 24)) % 4], 8) ^ arithmeticRightShift((u32)Sbox[((t1 >> 16) & 0xff) / 4][warpThreadIndex][((t1 >> 16)) % 4], 16) ^ arithmeticRightShift((u32)Sbox[((t2 >> 8) & 0xFF) / 4][warpThreadIndex][((t2 >> 8)) % 4], 24) ^ ((u32)Sbox[((t3 & 0xFF) / 4)][warpThreadIndex][((t3 & 0xFF) % 4)]) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = arithmeticRightShift((u32)Sbox[((t1 >> 24)) / 4][warpThreadIndex][((t1 >> 24)) % 4], 8) ^ arithmeticRightShift((u32)Sbox[((t2 >> 16) & 0xff) / 4][warpThreadIndex][((t2 >> 16)) % 4], 16) ^ arithmeticRightShift((u32)Sbox[((t3 >> 8) & 0xFF) / 4][warpThreadIndex][((t3 >> 8)) % 4], 24) ^ ((u32)Sbox[((t0 & 0xFF) / 4)][warpThreadIndex][((t0 & 0xFF) % 4)]) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = arithmeticRightShift((u32)Sbox[((t2 >> 24)) / 4][warpThreadIndex][((t2 >> 24)) % 4], 8) ^ arithmeticRightShift((u32)Sbox[((t3 >> 16) & 0xff) / 4][warpThreadIndex][((t3 >> 16)) % 4], 16) ^ arithmeticRightShift((u32)Sbox[((t0 >> 8) & 0xFF) / 4][warpThreadIndex][((t0 >> 8)) % 4], 24) ^ ((u32)Sbox[((t1 & 0xFF) / 4)][warpThreadIndex][((t1 & 0xFF) % 4)]) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = arithmeticRightShift((u32)Sbox[((t3 >> 24)) / 4][warpThreadIndex][((t3 >> 24)) % 4], 8) ^ arithmeticRightShift((u32)Sbox[((t0 >> 16) & 0xff) / 4][warpThreadIndex][((t0 >> 16)) % 4], 16) ^ arithmeticRightShift((u32)Sbox[((t1 >> 8) & 0xFF) / 4][warpThreadIndex][((t1 >> 8)) % 4], 24) ^ ((u32)Sbox[((t2 & 0xFF) / 4)][warpThreadIndex][((t2 & 0xFF) % 4)]) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
						printf("-------------------------------\n");
					}
				}
			}
		}
		// Overflow
		if (rk3Init == MAX_U32) { rk2Init++; }
		rk3Init++;		// Create key as 32 bit unsigned integers
	}
}
__global__ void AES128_CTR(u32* pt, u32* rk, u32* t0G, u32* t4G, u64* range, u8* SAES) {
	u64 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u8 Sbox[64][32][4];
	//	__shared__ u32 t4S[TABLE_SIZE][S_BOX_BANK_SIZE];
	__shared__ u32 rkS[AES_128_KEY_SIZE_INT];
	if (threadIdx.x < TABLE_SIZE) {
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
			Sbox[threadIdx.x / 4][bankIndex][threadIdx.x % 4] = SAES[threadIdx.x];
		}
		//		for (u8 bankIndex = 0; bankIndex < S_BOX_BANK_SIZE; bankIndex++) {	t4S[threadIdx.x][bankIndex] = t4G[threadIdx.x];	}
		//		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) { Sbox[threadIdx.x / 4][bankIndex][threadIdx.x % 4] = SAES[threadIdx.x]; }
		if (threadIdx.x < AES_128_KEY_SIZE_INT) { rkS[threadIdx.x] = rk[threadIdx.x]; }
	}
	__syncthreads();
	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	u32 s0, s1, s2, s3;
	pt0Init = pt[0];	pt1Init = pt[1];	pt2Init = pt[2];	pt3Init = pt[3];
	u64 threadRange = *range;
	u64 threadRangeStart = pt2Init;
	threadRangeStart = threadRangeStart << 32;
	threadRangeStart ^= pt3Init;
	threadRangeStart += threadIndex * threadRange;
	pt2Init = threadRangeStart >> 32;
	pt3Init = threadRangeStart & 0xFFFFFFFF;
	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {
		// Create plaintext as 32 bit unsigned integers
		s0 = pt0Init;		s1 = pt1Init;		s2 = pt2Init;		s3 = pt3Init;
		// First round just XORs input with key.
		s0 = s0 ^ rkS[0];		s1 = s1 ^ rkS[1];		s2 = s2 ^ rkS[2];		s3 = s3 ^ rkS[3];
		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {
			// Table based round function
			u32 rkStart = roundCount * 4 + 4;
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s3 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart];
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s0 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 1];
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s1 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 2];
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s2 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 3];
			s0 = t0;			s1 = t1;			s2 = t2;			s3 = t3;
		}
		// Calculate the last round key
		// Last round uses s-box directly and XORs to produce output.
/*		s0 = (t4S[t0 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t3) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[40];
		s1 = (t4S[t1 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t0) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[41];
		s2 = (t4S[t2 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t1) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[42];
		s3 = (t4S[t3 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t2) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[43];*/
		s0 = arithmeticRightShift((u64)Sbox[((t0 >> 24)) / 4][warpThreadIndex][((t0 >> 24)) % 4], 8) ^ arithmeticRightShift((u64)Sbox[((t1 >> 16) & 0xff) / 4][warpThreadIndex][((t1 >> 16)) % 4], 16) ^ arithmeticRightShift((u64)Sbox[((t2 >> 8) & 0xFF) / 4][warpThreadIndex][((t2 >> 8)) % 4], 24) ^ ((u64)Sbox[((t3 & 0xFF) / 4)][warpThreadIndex][((t3 & 0xFF) % 4)]) ^ rkS[40];
		s1 = arithmeticRightShift((u64)Sbox[((t1 >> 24)) / 4][warpThreadIndex][((t1 >> 24)) % 4], 8) ^ arithmeticRightShift((u64)Sbox[((t2 >> 16) & 0xff) / 4][warpThreadIndex][((t2 >> 16)) % 4], 16) ^ arithmeticRightShift((u64)Sbox[((t3 >> 8) & 0xFF) / 4][warpThreadIndex][((t3 >> 8)) % 4], 24) ^ ((u64)Sbox[((t0 & 0xFF) / 4)][warpThreadIndex][((t0 & 0xFF) % 4)]) ^ rkS[41];
		s2 = arithmeticRightShift((u64)Sbox[((t2 >> 24)) / 4][warpThreadIndex][((t2 >> 24)) % 4], 8) ^ arithmeticRightShift((u64)Sbox[((t3 >> 16) & 0xff) / 4][warpThreadIndex][((t3 >> 16)) % 4], 16) ^ arithmeticRightShift((u64)Sbox[((t0 >> 8) & 0xFF) / 4][warpThreadIndex][((t0 >> 8)) % 4], 24) ^ ((u64)Sbox[((t1 & 0xFF) / 4)][warpThreadIndex][((t1 & 0xFF) % 4)]) ^ rkS[42];
		s3 = arithmeticRightShift((u64)Sbox[((t3 >> 24)) / 4][warpThreadIndex][((t3 >> 24)) % 4], 8) ^ arithmeticRightShift((u64)Sbox[((t0 >> 16) & 0xff) / 4][warpThreadIndex][((t0 >> 16)) % 4], 16) ^ arithmeticRightShift((u64)Sbox[((t1 >> 8) & 0xFF) / 4][warpThreadIndex][((t1 >> 8)) % 4], 24) ^ ((u64)Sbox[((t2 & 0xFF) / 4)][warpThreadIndex][((t2 & 0xFF) % 4)]) ^ rkS[43];
		// Overflow
		if (pt3Init == MAX_U32) { pt2Init++; }
		pt3Init++;
	}
	if (threadIndex == 1048575) {
		printf("threadIndex : %lld\n", threadIndex);
		printf("Plaintext   : %08x %08x %08x %08x\n", pt0Init, pt1Init, pt2Init, pt3Init);
		printf("Ciphertext  : %08x %08x %08x %08x\n", s0, s1, s2, s3);
		printf("-------------------------------\n");
	}
}
__global__ void AES256_CTR(u32* pt, u32* rk, u32* t0G, u32* t4G, u64* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;
	int warpThreadIndexSBox = warpThreadIndex % S_BOX_BANK_SIZE;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S[TABLE_SIZE][S_BOX_BANK_SIZE];
	__shared__ u32 rkS[AES_256_KEY_SIZE_INT];

	if (threadIdx.x < TABLE_SIZE) {
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		for (u8 bankIndex = 0; bankIndex < S_BOX_BANK_SIZE; bankIndex++) {
			t4S[threadIdx.x][bankIndex] = t4G[threadIdx.x];
		}

		if (threadIdx.x < AES_256_KEY_SIZE_INT) {
			rkS[threadIdx.x] = rk[threadIdx.x];
		}

	}
	// </SHARED MEMORY>

	// Wait until every thread is ready
	__syncthreads();

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	u32 s0, s1, s2, s3;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = pt2Init;
	threadRangeStart = threadRangeStart << 32;
	threadRangeStart ^= pt3Init;
	threadRangeStart += (u64)threadIndex * threadRange;
	pt2Init = threadRangeStart >> 32;
	pt3Init = threadRangeStart & 0xFFFFFFFF;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		// Create plaintext as 32 bit unsigned integers
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rkS[0];
		s1 = s1 ^ rkS[1];
		s2 = s2 ^ rkS[2];
		s3 = s3 ^ rkS[3];

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1_256; roundCount++) {

			// Table based round function
			u32 rkStart = roundCount * 4 + 4;
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart];
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 1];
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 2];
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 3];

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t3) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[56];
		s1 = (t4S[t1 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t0) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[57];
		s2 = (t4S[t2 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t1) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[58];
		s3 = (t4S[t3 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t2) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[59];

		//if (threadIndex == 0 && rangeCount == 0) {
		//printf("Ciphertext : %08x %08x %08x %08x\n", s0, s1, s2, s3);
		//}

		// Overflow
		if (pt3Init == MAX_U32) {
			pt2Init++;
		}

		// Create key as 32 bit unsigned integers
		pt3Init++;
	}

	if (threadIndex == 1048575) {
		printf("Plaintext : %08x %08x %08x %08x\n", pt0Init, pt1Init, pt2Init, pt3Init);
		printf("Ciphertext : %08x %08x %08x %08x\n", s0, s1, s2, s3);
		printf("-------------------------------\n");
	}

}
__global__ void Wider_AES256_CTR(u32* pt, u32* rk, u32* t0G, u32* t4G, u64* range) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;
	int warpThreadIndexSBox = warpThreadIndex % S_BOX_BANK_SIZE;
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S[TABLE_SIZE][S_BOX_BANK_SIZE];
	__shared__ u32 rkS[120];
	if (threadIdx.x < TABLE_SIZE) {
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];		}
		for (u8 bankIndex = 0; bankIndex < S_BOX_BANK_SIZE; bankIndex++) {			t4S[threadIdx.x][bankIndex] = t4G[threadIdx.x];		}
		if (threadIdx.x < 120) {			rkS[threadIdx.x] = rk[threadIdx.x];		}
	}
	__syncthreads();
	u32 pt0Init, pt1Init, pt2Init, pt3Init, pt4Init, pt5Init, pt6Init, pt7Init;
	u32 s0, s1, s2, s3, s4, s5, s6, s7;
	pt0Init = pt[0];	pt1Init = pt[1];	pt2Init = pt[2];	pt3Init = pt[3]; pt4Init = pt[4];	pt5Init = pt[5];	pt6Init = pt[6];	pt7Init = pt[7];
	u32 threadRange = *range;
	u64 threadRangeStart = pt2Init;
	threadRangeStart = threadRangeStart << 32;
	threadRangeStart ^= pt3Init;
	threadRangeStart += (u64)threadIndex * threadRange;
	pt2Init = threadRangeStart >> 32;
	pt3Init = threadRangeStart & 0xFFFFFFFF;
	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {
		s0 = pt0Init;		s1 = pt1Init;		s2 = pt2Init;		s3 = pt3Init; s4 = pt4Init;		s5 = pt5Init;		s6 = pt6Init;		s7 = pt7Init;
		s0 = s0 ^ rkS[0];		s1 = s1 ^ rkS[1];		s2 = s2 ^ rkS[2];		s3 = s3 ^ rkS[3]; s4 = s4 ^ rkS[4];		s5 = s5 ^ rkS[5];		s6 = s6 ^ rkS[6];		s7 = s7 ^ rkS[7];
		u32 t0, t1, t2, t3, t4, t5, t6, t7;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1_256; roundCount++) {
			// Table based round function
			u32 rkStart = roundCount * 8 + 8;
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s4 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart];
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s4 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s5 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 1];
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s5 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s6 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 2];
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s4 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s6 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s7 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 3];
			t4 = t0S[s4 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s5 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s7 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s0 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 4];
			t5 = t0S[s5 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s6 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s1 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 5];
			t6 = t0S[s6 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s7 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s2 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 6];
			t7 = t0S[s7 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s3 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 7];
			s0 = t0;			s1 = t1;			s2 = t2;			s3 = t3; s4 = t4;			s5 = t5;			s6 = t6;			s7 = t7;
		}
		// Calculate the last round key
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t4) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[112];
		s1 = (t4S[t1 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t4 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t5) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[113];
		s2 = (t4S[t2 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t5 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t6) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[114];
		s3 = (t4S[t3 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t4 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t6 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t7) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[115];
		s4 = (t4S[t4 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t5 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t7 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t0) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[116];
		s5 = (t4S[t5 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t7 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t1) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[117];
		s6 = (t4S[t6 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t7 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t2) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[118];
		s7 = (t4S[t7 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t3) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[119];

		// Overflow
		if (pt3Init == MAX_U32) {			pt2Init++;		}
		// Create key as 32 bit unsigned integers
		pt3Init++;
	}
	if (threadIndex == 1048575) {
		printf("Plaintext : %08x %08x %08x %08x\n", pt0Init, pt1Init, pt2Init, pt3Init);
		printf("Ciphertext : %08x %08x %08x %08x\n", s0, s1, s2, s3);
		printf("-------------------------------\n");
	}

}
__global__ void Wider_AES128_CTR(u32* pt, u32* rk, u32* t0G, u32* t4G, u64* range, u8* SAES) {
	u64 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u8 Sbox[64][32][4];
	__shared__ u32 rkS[WAES_256_KEY_SIZE_INT];
	if (threadIdx.x < TABLE_SIZE) {
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
			Sbox[threadIdx.x / 4][bankIndex][threadIdx.x % 4] = SAES[threadIdx.x];
		}
		if (threadIdx.x < WAES_256_KEY_SIZE_INT) { rkS[threadIdx.x] = rk[threadIdx.x]; }
	}
	__syncthreads();
	u32 pt0Init, pt1Init, pt2Init, pt3Init, pt4Init, pt5Init, pt6Init, pt7Init;
	u32 s0, s1, s2, s3, s4, s5, s6, s7;
	pt0Init = pt[0];	pt1Init = pt[1];	pt2Init = pt[2];	pt3Init = pt[3]; pt4Init = pt[4];	pt5Init = pt[5];	pt6Init = pt[6];	pt7Init = pt[7];
	u64 threadRange = *range;
	u64 threadRangeStart = pt2Init;
	threadRangeStart = threadRangeStart << 32;
	threadRangeStart ^= pt3Init;
	threadRangeStart += threadIndex * threadRange;
	pt2Init = threadRangeStart >> 32;
	pt3Init = threadRangeStart & 0xFFFFFFFF;
	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {
		// Create plaintext as 32 bit unsigned integers
		s0 = pt0Init;		s1 = pt1Init;		s2 = pt2Init;		s3 = pt3Init; s4 = pt4Init;		s5 = pt5Init;		s6 = pt6Init;		s7 = pt7Init;
		// First round just XORs input with key.
		s0 = s0 ^ rkS[0];		s1 = s1 ^ rkS[1];		s2 = s2 ^ rkS[2];		s3 = s3 ^ rkS[3]; s4 = s4 ^ rkS[4];		s5 = s5 ^ rkS[5];		s6 = s6 ^ rkS[6];		s7 = s7 ^ rkS[7];
		u32 t0, t1, t2, t3, t4, t5, t6, t7;
		for (u8 roundCount = 0; roundCount < 13; roundCount++) {
			// Table based round function
			u32 rkStart = roundCount * 8 + 8;
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s4 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart];
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s4 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s5 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 1];
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s5 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s6 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 2];
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s4 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s6 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s7 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 3];
			t4 = t0S[s4 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s5 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s7 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s0 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 4];
			t5 = t0S[s5 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s6 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s1 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 5];
			t6 = t0S[s6 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s7 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s2 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 6];
			t7 = t0S[s7 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s3 & 0xFF][warpThreadIndex], 24) ^ rkS[rkStart + 7];
			s0 = t0;			s1 = t1;			s2 = t2;			s3 = t3; s4 = t4;			s5 = t5;			s6 = t6;			s7 = t7;
		}
		// Calculate the last round key
		// Last round uses s-box directly and XORs to produce output.
		s0 = arithmeticRightShift((u64)Sbox[((t0 >> 24)) / 4][warpThreadIndex][((t0 >> 24)) % 4], 8) ^ arithmeticRightShift((u64)Sbox[((t1 >> 16) & 0xff) / 4][warpThreadIndex][((t1 >> 16)) % 4], 16) ^ arithmeticRightShift((u64)Sbox[((t3 >> 8) & 0xFF) / 4][warpThreadIndex][((t3 >> 8)) % 4], 24) ^ ((u64)Sbox[((t4 & 0xFF) / 4)][warpThreadIndex][((t4 & 0xFF) % 4)]) ^ rkS[112];
		s1 = arithmeticRightShift((u64)Sbox[((t1 >> 24)) / 4][warpThreadIndex][((t1 >> 24)) % 4], 8) ^ arithmeticRightShift((u64)Sbox[((t2 >> 16) & 0xff) / 4][warpThreadIndex][((t2 >> 16)) % 4], 16) ^ arithmeticRightShift((u64)Sbox[((t4 >> 8) & 0xFF) / 4][warpThreadIndex][((t4 >> 8)) % 4], 24) ^ ((u64)Sbox[((t5 & 0xFF) / 4)][warpThreadIndex][((t5 & 0xFF) % 4)]) ^ rkS[113];
		s2 = arithmeticRightShift((u64)Sbox[((t2 >> 24)) / 4][warpThreadIndex][((t2 >> 24)) % 4], 8) ^ arithmeticRightShift((u64)Sbox[((t3 >> 16) & 0xff) / 4][warpThreadIndex][((t3 >> 16)) % 4], 16) ^ arithmeticRightShift((u64)Sbox[((t5 >> 8) & 0xFF) / 4][warpThreadIndex][((t5 >> 8)) % 4], 24) ^ ((u64)Sbox[((t6 & 0xFF) / 4)][warpThreadIndex][((t6 & 0xFF) % 4)]) ^ rkS[114];
		s3 = arithmeticRightShift((u64)Sbox[((t3 >> 24)) / 4][warpThreadIndex][((t3 >> 24)) % 4], 8) ^ arithmeticRightShift((u64)Sbox[((t4 >> 16) & 0xff) / 4][warpThreadIndex][((t4 >> 16)) % 4], 16) ^ arithmeticRightShift((u64)Sbox[((t6 >> 8) & 0xFF) / 4][warpThreadIndex][((t6 >> 8)) % 4], 24) ^ ((u64)Sbox[((t7 & 0xFF) / 4)][warpThreadIndex][((t7 & 0xFF) % 4)]) ^ rkS[115];
		s4 = arithmeticRightShift((u64)Sbox[((t4 >> 24)) / 4][warpThreadIndex][((t4 >> 24)) % 4], 8) ^ arithmeticRightShift((u64)Sbox[((t5 >> 16) & 0xff) / 4][warpThreadIndex][((t5 >> 16)) % 4], 16) ^ arithmeticRightShift((u64)Sbox[((t7 >> 8) & 0xFF) / 4][warpThreadIndex][((t7 >> 8)) % 4], 24) ^ ((u64)Sbox[((t0 & 0xFF) / 4)][warpThreadIndex][((t0 & 0xFF) % 4)]) ^ rkS[116];
		s5 = arithmeticRightShift((u64)Sbox[((t5 >> 24)) / 4][warpThreadIndex][((t5 >> 24)) % 4], 8) ^ arithmeticRightShift((u64)Sbox[((t6 >> 16) & 0xff) / 4][warpThreadIndex][((t6 >> 16)) % 4], 16) ^ arithmeticRightShift((u64)Sbox[((t0 >> 8) & 0xFF) / 4][warpThreadIndex][((t0 >> 8)) % 4], 24) ^ ((u64)Sbox[((t1 & 0xFF) / 4)][warpThreadIndex][((t1 & 0xFF) % 4)]) ^ rkS[117];
		s6 = arithmeticRightShift((u64)Sbox[((t6 >> 24)) / 4][warpThreadIndex][((t6 >> 24)) % 4], 8) ^ arithmeticRightShift((u64)Sbox[((t7 >> 16) & 0xff) / 4][warpThreadIndex][((t7 >> 16)) % 4], 16) ^ arithmeticRightShift((u64)Sbox[((t1 >> 8) & 0xFF) / 4][warpThreadIndex][((t1 >> 8)) % 4], 24) ^ ((u64)Sbox[((t2 & 0xFF) / 4)][warpThreadIndex][((t2 & 0xFF) % 4)]) ^ rkS[118];
		s7 = arithmeticRightShift((u64)Sbox[((t7 >> 24)) / 4][warpThreadIndex][((t7 >> 24)) % 4], 8) ^ arithmeticRightShift((u64)Sbox[((t0 >> 16) & 0xff) / 4][warpThreadIndex][((t0 >> 16)) % 4], 16) ^ arithmeticRightShift((u64)Sbox[((t2 >> 8) & 0xFF) / 4][warpThreadIndex][((t2 >> 8)) % 4], 24) ^ ((u64)Sbox[((t3 & 0xFF) / 4)][warpThreadIndex][((t3 & 0xFF) % 4)]) ^ rkS[119];
		// Overflow
		if (pt3Init == MAX_U32) { pt2Init++; }
		pt3Init++;
	}
	if (threadIndex == 1048575) {
		printf("threadIndex : %lld\n", threadIndex);
		printf("Plaintext   : %08x %08x %08x %08x %08x %08x %08x %08x\n", pt0Init, pt1Init, pt2Init, pt3Init, pt4Init, pt5Init, pt6Init, pt7Init);
		printf("Ciphertext  : %08x %08x %08x %08x %08x %08x %08x %08x\n", s0, s1, s2, s3, s4, s5, s6, s7);
		printf("-------------------------------\n");
	}
}
__host__ void keyExpansion(u32* key, u32* rk) {
	u32 rk0, rk1, rk2, rk3;
	rk0 = key[0];
	rk1 = key[1];
	rk2 = key[2];
	rk3 = key[3];
	rk[0] = rk0;
	rk[1] = rk1;
	rk[2] = rk2;
	rk[3] = rk3;
	for (u8 roundCount = 0; roundCount < ROUND_COUNT; roundCount++) {
		u32 temp = rk3;
		rk0 = rk0 ^ T4_3[(temp >> 16) & 0xff] ^ T4_2[(temp >> 8) & 0xff] ^ T4_1[(temp) & 0xff] ^ T4_0[(temp >> 24)] ^ RCON32[roundCount];
		rk1 = rk1 ^ rk0;
		rk2 = rk2 ^ rk1;
		rk3 = rk2 ^ rk3;
		rk[roundCount * 4 + 4] = rk0;
		rk[roundCount * 4 + 5] = rk1;
		rk[roundCount * 4 + 6] = rk2;
		rk[roundCount * 4 + 7] = rk3;
	}
}
__host__ void keyExpansion256(u32* key, u32* rk) {

	u32 rk0, rk1, rk2, rk3, rk4, rk5, rk6, rk7;
	rk0 = key[0];
	rk1 = key[1];
	rk2 = key[2];
	rk3 = key[3];
	rk4 = key[4];
	rk5 = key[5];
	rk6 = key[6];
	rk7 = key[7];

	rk[0] = rk0;
	rk[1] = rk1;
	rk[2] = rk2;
	rk[3] = rk3;
	rk[4] = rk4;
	rk[5] = rk5;
	rk[6] = rk6;
	rk[7] = rk7;

	for (u8 roundCount = 0; roundCount < ROUND_COUNT_256; roundCount++) {
		u32 temp = rk7;
		rk0 = rk0 ^ T4_3[(temp >> 16) & 0xff] ^ T4_2[(temp >> 8) & 0xff] ^ T4_1[(temp) & 0xff] ^ T4_0[(temp >> 24)] ^ RCON32[roundCount];
		rk1 = rk1 ^ rk0;
		rk2 = rk2 ^ rk1;
		rk3 = rk3 ^ rk2;
		rk4 = rk4 ^ T4_3[(rk3 >> 24) & 0xff] ^ T4_2[(rk3 >> 16) & 0xff] ^ T4_1[(rk3 >> 8) & 0xff] ^ T4_0[rk3 & 0xff];
		rk5 = rk5 ^ rk4;
		rk6 = rk6 ^ rk5;
		rk7 = rk7 ^ rk6;

		rk[roundCount * 8 + 8] = rk0;
		rk[roundCount * 8 + 9] = rk1;
		rk[roundCount * 8 + 10] = rk2;
		rk[roundCount * 8 + 11] = rk3;
		if (roundCount == 6) {
			break;
		}
		rk[roundCount * 8 + 12] = rk4;
		rk[roundCount * 8 + 13] = rk5;
		rk[roundCount * 8 + 14] = rk6;
		rk[roundCount * 8 + 15] = rk7;

	}

	//for (int i = 0; i < 60; i++) {
	//	printf("%08x ", rk[i]);
	//	if ((i + 1) % 4 == 0) {
	//		printf("Round: %d\n", i / 4);
	//	}
	//}
}
__host__ void keyExpansionW256(u32* key, u32* rk) {
	u32 rk0, rk1, rk2, rk3, rk4, rk5, rk6, rk7;
	rk0 = key[0];
	rk1 = key[1];
	rk2 = key[2];
	rk3 = key[3];
	rk4 = key[4];
	rk5 = key[5];
	rk6 = key[6];
	rk7 = key[7];

	rk[0] = rk0;
	rk[1] = rk1;
	rk[2] = rk2;
	rk[3] = rk3;
	rk[4] = rk4;
	rk[5] = rk5;
	rk[6] = rk6;
	rk[7] = rk7;

	for (u8 roundCount = 0; roundCount < ROUND_COUNT_256; roundCount++) {
		u32 temp = rk7;
		rk0 = rk0 ^ T4_3[(temp >> 16) & 0xff] ^ T4_2[(temp >> 8) & 0xff] ^ T4_1[(temp) & 0xff] ^ T4_0[(temp >> 24)] ^ RCON32[roundCount];
		rk1 = rk1 ^ rk0;
		rk2 = rk2 ^ rk1;
		rk3 = rk3 ^ rk2;
		rk4 = rk4 ^ T4_3[(rk3 >> 24) & 0xff] ^ T4_2[(rk3 >> 16) & 0xff] ^ T4_1[(rk3 >> 8) & 0xff] ^ T4_0[rk3 & 0xff];
		rk5 = rk5 ^ rk4;
		rk6 = rk6 ^ rk5;
		rk7 = rk7 ^ rk6;

		rk[roundCount * 8 + 8] = rk0;
		rk[roundCount * 8 + 9] = rk1;
		rk[roundCount * 8 + 10] = rk2;
		rk[roundCount * 8 + 11] = rk3;
		if (roundCount == 6) {
			break;
		}
		rk[roundCount * 8 + 12] = rk4;
		rk[roundCount * 8 + 13] = rk5;
		rk[roundCount * 8 + 14] = rk6;
		rk[roundCount * 8 + 15] = rk7;

	}
}
__host__ int AES128ExhaustiveSearch(int choice) {
	printf("\n");	printf("########## AES-128 Exhaustive Search Implementation ##########\n");	printf("\n");
	// Allocate plaintext, ciphertext and initial round key
	u32* pt, * ct, * rk;
	gpuErrorCheck(hipMallocManaged(&pt, 4 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&ct, 4 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&rk, 4 * sizeof(u32)));
	pt[0] = 0x3243F6A8U;	pt[1] = 0x885A308DU;	pt[2] = 0x313198A2U;	pt[3] = 0xE0370734U;
	//	pt[0] = 0;	pt[1] = 0;	pt[2] = 0;	pt[3] = 0;
	ct[0] = 0x3925841DU;	ct[1] = 0x02DC09FBU;	ct[2] = 0xDC118597U;	ct[3] = 0x196A0B32U;
	// aes-cipher-internals.xlsx
	rk[0] = 0x2B7E1516U;	rk[1] = 0x28AED2A6U;	rk[2] = 0xABF71588U;	rk[3] = 0x09CF0000U;
	// Allocate RCON values
	u32* rcon;
	gpuErrorCheck(hipMallocManaged(&rcon, RCON_SIZE * sizeof(u32)));
	for (int i = 0; i < RCON_SIZE; i++) { rcon[i] = RCON32[i]; }
	// Allocate Tables
	u32* t0, * t1, * t2, * t3, * t4, * t4_0, * t4_1, * t4_2, * t4_3;
	u8* SAES_d; // Cihangir
	gpuErrorCheck(hipMallocManaged(&t0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t3, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_3, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&SAES_d, 256 * sizeof(u8))); // Cihangir
	for (int i = 0; i < TABLE_SIZE; i++) {
		t0[i] = T0[i];		t1[i] = T1[i];		t2[i] = T2[i];		t3[i] = T3[i];		t4[i] = T4[i];
		t4_0[i] = T4_0[i];		t4_1[i] = T4_1[i];		t4_2[i] = T4_2[i];		t4_3[i] = T4_3[i];
	}
	for (int i = 0; i < 256; i++) SAES_d[i] = SAES[i]; // Cihangir
	printf("-------------------------------\n");
	u64* range = calculateRange();
	/*	printf("Plaintext                     : %08x %08x %08x %08x\n", pt[0], pt[1], pt[2], pt[3]);
		printf("Ciphertext                    : %08x %08x %08x %08x\n", ct[0], ct[1], ct[2], ct[3]);
		printf("Initial Key                   : %08x %08x %08x %08x\n", rk[0], rk[1], rk[2], rk[3]);
		printf("-------------------------------\n");*/

	clock_t beginTime = clock();
	if (choice == 1) AES128_Exhaustive_Search << <BLOCKS, THREADS >> > (pt, ct, rk, t0, t4, rcon, range, SAES_d);
	hipDeviceSynchronize();
	printf("Time elapsed: %f sec\n", float(clock() - beginTime) / CLOCKS_PER_SEC);

	printf("-------------------------------\n");
	printLastCUDAError();
	// Free alocated arrays
	hipFree(range); hipFree(pt);	hipFree(ct);	hipFree(rk);	hipFree(t0);	hipFree(t1);	hipFree(t2);	hipFree(t3);	hipFree(t4);
	hipFree(t4_0);	hipFree(t4_1);	hipFree(t4_2);	hipFree(t4_3);	hipFree(rcon); hipFree(SAES_d);
	return 0;
}
__host__ int AES128Ctr() {
	printf("\n");
	printf("########## AES-128 Counter Mode Implementation ##########\n");
	printf("\n");

	// Allocate plaintext and every round key
	u32* pt, * rk, * roundKeys;
	gpuErrorCheck(hipMallocManaged(&pt, 4 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&rk, 4 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&roundKeys, AES_128_KEY_SIZE_INT * sizeof(u32)));

	pt[0] = 0x3243F6A8U;
	pt[1] = 0x885A308DU;
	pt[2] = 0x313198A2U;
	pt[3] = 0x00000000U;

	rk[0] = 0x2B7E1516U;
	rk[1] = 0x28AED2A6U;
	rk[2] = 0xABF71588U;
	rk[3] = 0x09CF4F3CU;

	// Allocate RCON values
	u32* rcon;
	gpuErrorCheck(hipMallocManaged(&rcon, RCON_SIZE * sizeof(u32)));
	for (int i = 0; i < RCON_SIZE; i++) {
		rcon[i] = RCON32[i];
	}

	// Allocate Tables
	u32* t0, * t1, * t2, * t3, * t4, * t4_0, * t4_1, * t4_2, * t4_3;
	u8* SAES_d; // Cihangir
	gpuErrorCheck(hipMallocManaged(&t0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t3, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_3, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&SAES_d, 256 * sizeof(u8))); // Cihangir
	for (int i = 0; i < TABLE_SIZE; i++) {
		t0[i] = T0[i];
		t1[i] = T1[i];
		t2[i] = T2[i];
		t3[i] = T3[i];
		t4[i] = T4[i];
		t4_0[i] = T4_0[i];
		t4_1[i] = T4_1[i];
		t4_2[i] = T4_2[i];
		t4_3[i] = T4_3[i];
	}
	for (int i = 0; i < 256; i++) SAES_d[i] = SAES[i]; // Cihangir
	printf("-------------------------------\n");
	u64* range = calculateRange();
	/*	printf("Initial Plaintext              : %08x %08x %08x %08x\n", pt[0], pt[1], pt[2], pt[3]);
		printf("Initial Key                    : %08x %08x %08x %08x\n", rk[0], rk[1], rk[2], rk[3]);
		printf("-------------------------------\n");*/

		// Key expansion
	keyExpansion(rk, roundKeys);

	clock_t beginTime = clock();
	// Kernels
//	counterWithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox<<<BLOCKS, THREADS>>>(pt, roundKeys, t0, t4, range);
	AES128_CTR << <BLOCKS, THREADS >> > (pt, roundKeys, t0, t4, range, SAES_d);
	//	counterWithOneTableExtendedSharedMemoryBytePerm4ShiftedSbox<<<BLOCKS, THREADS>>>(pt, roundKeys, t0, t4_0, t4_1, t4_2, t4_3, range);
	//	hipMemcpy(rk, pt, 4*sizeof(u32), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	printf("Time elapsed: %f sec\n", float(clock() - beginTime) / CLOCKS_PER_SEC);
	printf("-------------------------------\n");
	printLastCUDAError();
	printf("plaintext: %x %x %x %x\n", rk[0], rk[1], rk[2], rk[3]);

	// Free alocated arrays
	hipFree(range);
	hipFree(pt);
	hipFree(rk);
	hipFree(roundKeys);
	hipFree(t0);
	hipFree(t1);
	hipFree(t2);
	hipFree(t3);
	hipFree(t4);
	hipFree(t4_0);
	hipFree(t4_1);
	hipFree(t4_2);
	hipFree(t4_3);
	hipFree(rcon);

	return 0;
}
__host__ int WAES128Ctr() {
	printf("\n");	printf("########## WAES-128 Counter Mode Implementation ##########\n");	printf("\n");
	// Allocate plaintext and every round key
	u32* pt, * rk, * roundKeys;
	gpuErrorCheck(hipMallocManaged(&pt, 8 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&rk, 8 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&roundKeys, WAES_256_KEY_SIZE_INT * sizeof(u32)));
	pt[0] = 0x3243F6A8U;	pt[1] = 0x885A308DU;	pt[2] = 0x313198A2U;	pt[3] = 0x00000000U;	pt[4] = 0x3243F6A8U;	pt[5] = 0x885A308DU;	pt[6] = 0x313198A2U;	pt[7] = 0x00000000U;
	rk[0] = 0x2B7E1516U;	rk[1] = 0x28AED2A6U;	rk[2] = 0xABF71588U;	rk[3] = 0x09CF4F3CU;	rk[4] = 0x2B7E1516U;	rk[5] = 0x28AED2A6U;	rk[6] = 0xABF71588U;	rk[7] = 0x09CF4F3CU;
	// Allocate RCON values
	u32* rcon;
	gpuErrorCheck(hipMallocManaged(&rcon, RCON_SIZE * 2* sizeof(u32)));
	for (int i = 0; i < RCON_SIZE*2; i++) {
		rcon[i] = RCON32[i];
	}
	// Allocate Tables
	u32* t0, * t1, * t2, * t3, * t4, * t4_0, * t4_1, * t4_2, * t4_3;
	u8* SAES_d; // Cihangir
	gpuErrorCheck(hipMallocManaged(&t0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t3, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_3, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&SAES_d, 256 * sizeof(u8))); // Cihangir
	for (int i = 0; i < TABLE_SIZE; i++) {
		t0[i] = T0[i];
		t1[i] = T1[i];
		t2[i] = T2[i];
		t3[i] = T3[i];
		t4[i] = T4[i];
		t4_0[i] = T4_0[i];
		t4_1[i] = T4_1[i];
		t4_2[i] = T4_2[i];
		t4_3[i] = T4_3[i];
	}
	for (int i = 0; i < 256; i++) SAES_d[i] = SAES[i]; // Cihangir
	printf("-------------------------------\n");
	u64* range = calculateRange();
	keyExpansionW256(rk, roundKeys);	
	clock_t beginTime = clock();
	Wider_AES128_CTR << <BLOCKS, THREADS >> > (pt, roundKeys, t0, t4, range, SAES_d);
	hipDeviceSynchronize();
	printf("Time elapsed: %f sec\n", float(clock() - beginTime) / CLOCKS_PER_SEC);
	printf("-------------------------------\n");
	printLastCUDAError();
	printf("plaintext: %x %x %x %x\n", rk[0], rk[1], rk[2], rk[3]);

	hipFree(range);	hipFree(pt);	hipFree(rk);	hipFree(roundKeys);	hipFree(t0);	hipFree(t1);	hipFree(t2);	hipFree(t3);	hipFree(t4);	hipFree(t4_0);	hipFree(t4_1);	hipFree(t4_2);	hipFree(t4_3);	hipFree(rcon);
	return 0;
}
__host__ int AES256Ctr() {
	printf("\n");
	printf("########## AES-256 Counter Mode Implementation ##########\n");
	printf("\n");

	// Allocate plaintext and every round key
	u32* pt, * ct, * rk256, * roundKeys256;
	gpuErrorCheck(hipMallocManaged(&pt, 4 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&ct, 4 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&rk256, 8 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&roundKeys256, AES_256_KEY_SIZE_INT * sizeof(u32)));

	pt[0] = 0x6bc1bee2U;
	pt[1] = 0x2e409f96U;
	pt[2] = 0xe93d7e11U;
	pt[3] = 0x7393172aU;

	ct[0] = 0xF3EED1BDU;
	ct[1] = 0xB5D2A03CU;
	ct[2] = 0x064B5A7EU;
	ct[3] = 0x3DB181F8U;

	rk256[0] = 0x603deb10U;
	rk256[1] = 0x15ca71beU;
	rk256[2] = 0x2b73aef0U;
	rk256[3] = 0x857d7781U;
	rk256[4] = 0x1f352c07U;
	rk256[5] = 0x3b6108d7U;
	rk256[6] = 0x2d9810a3U;
	rk256[7] = 0x0914dff4U;

	// Allocate RCON values
	u32* rcon;
	gpuErrorCheck(hipMallocManaged(&rcon, RCON_SIZE * sizeof(u32)));
	for (int i = 0; i < RCON_SIZE; i++) {
		rcon[i] = RCON32[i];
	}

	// Allocate Tables
	u32* t0, * t1, * t2, * t3, * t4, * t4_0, * t4_1, * t4_2, * t4_3;
	gpuErrorCheck(hipMallocManaged(&t0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t3, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_3, TABLE_SIZE * sizeof(u32)));
	for (int i = 0; i < TABLE_SIZE; i++) {
		t0[i] = T0[i];
		t1[i] = T1[i];
		t2[i] = T2[i];
		t3[i] = T3[i];
		t4[i] = T4[i];
		t4_0[i] = T4_0[i];
		t4_1[i] = T4_1[i];
		t4_2[i] = T4_2[i];
		t4_3[i] = T4_3[i];
	}

	printf("-------------------------------\n");
	u64* range = calculateRange();
	/*	printf("Initial Plaintext              : %08x %08x %08x %08x\n", pt[0], pt[1], pt[2], pt[3]);
		printf("Initial Key                    : %08x %08x %08x %08x %08x %08x %08x %08x\n", rk256[0], rk256[1], rk256[2], rk256[3], rk256[4], rk256[5], rk256[6], rk256[7]);
		printf("-------------------------------\n");*/

	keyExpansion256(rk256, roundKeys256);
	clock_t beginTime = clock();
	// Kernels
	AES256_CTR << <BLOCKS, THREADS >> > (pt, roundKeys256, t0, t4, range);

	hipDeviceSynchronize();
	printf("Time elapsed: %f sec\n", float(clock() - beginTime) / CLOCKS_PER_SEC);
	printf("-------------------------------\n");
	printLastCUDAError();

	// Free alocated arrays
	hipFree(range);
	hipFree(pt);
	hipFree(ct);
	hipFree(rk256);
	hipFree(roundKeys256);
	hipFree(t0);
	hipFree(t1);
	hipFree(t2);
	hipFree(t3);
	hipFree(t4);
	hipFree(t4_0);
	hipFree(t4_1);
	hipFree(t4_2);
	hipFree(t4_3);
	hipFree(rcon);


	return 0;
}
__host__ int WAES256Ctr() {
	printf("\n");	printf("########## WAES-256 Counter Mode Implementation ##########\n");	printf("\n");
	// Allocate plaintext and every round key
	u32* pt, * ct, * rk256, * roundKeys256;
	gpuErrorCheck(hipMallocManaged(&pt, 8 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&ct, 8 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&rk256, 8 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&roundKeys256, WAES_256_KEY_SIZE_INT * sizeof(u32)));
	pt[0] = 0x3243F6A8U;	pt[1] = 0x885A308DU;	pt[2] = 0x313198A2U;	pt[3] = 0x00000000U;	pt[4] = 0x3243F6A8U;	pt[5] = 0x885A308DU;	pt[6] = 0x313198A2U;	pt[7] = 0x00000000U;
	ct[0] = 0xF3EED1BDU;	ct[1] = 0xB5D2A03CU;	ct[2] = 0x064B5A7EU;	ct[3] = 0x3DB181F8U;
	rk256[0] = 0x603deb10U;	rk256[1] = 0x15ca71beU;	rk256[2] = 0x2b73aef0U;	rk256[3] = 0x857d7781U;	rk256[4] = 0x1f352c07U;	rk256[5] = 0x3b6108d7U;	rk256[6] = 0x2d9810a3U;	rk256[7] = 0x0914dff4U;
	// Allocate RCON values
	u32* rcon;
	gpuErrorCheck(hipMallocManaged(&rcon, RCON_SIZE *2* sizeof(u32)));
	for (int i = 0; i < RCON_SIZE; i++) {		rcon[i] = RCON32[i];	}
	// Allocate Tables
	u32* t0, * t1, * t2, * t3, * t4, * t4_0, * t4_1, * t4_2, * t4_3;
	gpuErrorCheck(hipMallocManaged(&t0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t3, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_3, TABLE_SIZE * sizeof(u32)));
	for (int i = 0; i < TABLE_SIZE; i++) {
		t0[i] = T0[i];
		t1[i] = T1[i];
		t2[i] = T2[i];
		t3[i] = T3[i];
		t4[i] = T4[i];
		t4_0[i] = T4_0[i];
		t4_1[i] = T4_1[i];
		t4_2[i] = T4_2[i];
		t4_3[i] = T4_3[i];
	}
	printf("-------------------------------\n");
	u64* range = calculateRange();
	keyExpansionW256(rk256, roundKeys256);
	clock_t beginTime = clock();
	Wider_AES256_CTR << <BLOCKS, THREADS >> > (pt, roundKeys256, t0, t4, range);
	hipDeviceSynchronize();
	printf("Time elapsed: %f sec\n", float(clock() - beginTime) / CLOCKS_PER_SEC);
	printf("-------------------------------\n");
	printLastCUDAError();
	hipFree(range);	hipFree(pt);	hipFree(ct);	hipFree(rk256);	hipFree(roundKeys256);	hipFree(t0);	hipFree(t1);	hipFree(t2);	hipFree(t3);	hipFree(t4);	hipFree(t4_0);	hipFree(t4_1);	hipFree(t4_2);	hipFree(t4_3);	hipFree(rcon);
	return 0;
}
int main() {
	hipSetDevice(0);
	int choice;
	printf(
		"(1)  AES-128 Exhaustive Search\n"
		"(2)  AES-128 CTR \n"
		"(3)  Wider AES-128 Exhaustive Search\n"
		"(4)  Wider AES-128/192/256 CTR \n"
		"(5)  AES-256 CTR \n"
		"(6)  WAES-256 CTR \n"
		"Choice: ");
	scanf_s("%d", &choice);
	if (choice == 1) AES128ExhaustiveSearch(1);
	else if (choice == 2) AES128Ctr();
	else if (choice == 4) WAES128Ctr();
	else if (choice == 5) AES256Ctr();
	else if (choice == 6) WAES256Ctr();
	else printf("Wrong selection\n");
	return 0;
}